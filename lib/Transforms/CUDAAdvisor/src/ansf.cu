#include "hip/hip_runtime.h"
#include<sys/mman.h>
#include<assert.h>
#include<iostream>
#include<string.h>
#include "../common.h"
#include "types.h"

__device__ int CTALB = 0; 			//the lower bound of CTA id you want to profile
__device__ int CTAUB = 99999;		//the upper bound of CTA id you want to profile 
__device__ int CONSTANCE = 128;
__device__ int aliveCTA = 0;
__device__ bool VERBOSE=false;
__device__ bool CALLPATHVERBOSE=false;

extern "C"
{	//so that no mangling for function names
	__device__ void takeString(void* , int);
	__device__ void RetKernel(void*);
	__device__ void passBasicBlock(int, int, int, int, void*);
	__device__ void print5(void*, int, int, int, int, void*);
	__device__ void print4(void*);
	__device__ void callFunc(void* , void* , int , int, void*);
	__device__ int getContextID(void*);
	__device__ void* InitKernel(void*);
	__device__ void print1(int);
}

__device__ unsigned long long ccnntt = 1;							//the very first element is reserved for metadata
__device__ unsigned long long bbccnntt = 1;							//the very first element is reserved for metadata

__device__ int* buffer_oN_DeViCe;									//should be multiples of 6
__device__ char funcDic[UNIQUE_FUNC_DEVICE][FUNC_NAME_LEN]; 		//maintains 100 unique functions and 31 chars for each
__device__ int dicHeight = 0;										// size of funcDic[][]

__device__ CallSite_t contextDic[TOTAL_NUMBER_CONTEXT][CALL_PATH_LEN_DEVICE]; //maintains 100 unique contexts, each has up to 10 function
__device__ int cHeight = 0;

__device__ void mystrcpy(char* dst, char* src)
{
	int cnt = 0;
	while ( src[cnt] != '\0' && cnt < FUNC_NAME_LEN-1) { //never exceeds this 30 limit
		dst[cnt] = src[cnt];
		cnt++;
	}
	dst[cnt] = '\0';
	return;
}

__device__ bool mystrcmp(char* dst, char* src)
{
	int cnt = 0;
	while ( cnt < FUNC_NAME_LEN-1 ) { //never exceeds this 30 limit
		if ( dst[cnt] == '\0' && src[cnt] == '\0')
			return true;
		if (dst[cnt] != src[cnt])
			return false;
		cnt++;
	}
	return true;
}

__device__ int getFuncID(char* func)
{
	if (dicHeight == 0 ) { //the very first function
		mystrcpy(funcDic[0], func);
		dicHeight ++;
		return 0;
	}

	for(int i=0; i < dicHeight; i++) {
		bool found = mystrcmp( funcDic[i],  func );
		if(found)
			return i;
	}
	//if you are here, means we have a new func
	mystrcpy(funcDic[dicHeight], func); 
	dicHeight ++;
	return dicHeight-1;
}

__device__ void updateCallStack(int caller, int callee, short sline, short scolm, int bid, int tid, void* p_stackzone)
{
	int offset = bid*blockDim.x*blockDim.y+tid;
	CallSite_t* callStack = (CallSite_t*) p_stackzone;
	int bytesPerThread = (CALL_PATH_LEN_DEVICE*sizeof(CallSite_t));
	int* temp = (int*)( (char*)p_stackzone + bytesPerThread+16);	//offset by 16 to be safe, need to be consistent
	int &height = *temp;		
	
	if (height==0) {
		callStack[0].id = caller;
		callStack[0].sline = sline;
		callStack[0].scolm = scolm;
		callStack[1].id = callee;
		callStack[1].sline = -1;
		callStack[1].scolm = -1;
		height=2;
		return;
	}

	int p_caller = callStack[height-2].id;
	int p_callee = callStack[height-1].id;
	if ( p_caller == caller && p_callee == callee) {       //repeated call
		callStack[height-2].sline = sline;
		callStack[height-2].scolm = scolm;
		return;
	} else if ( p_caller == caller && p_callee != callee) {       
		//the same parent called a different function, simply update the callee
		callStack[height-1].id = callee;
		callStack[height-2].sline = sline;
		callStack[height-2].scolm = scolm;
		return;
	} else if ( p_callee == caller) {       // a typical call path
		callStack[height-1].sline = sline;
		callStack[height-1].scolm = scolm;
		callStack[height].id = callee;
		callStack[height].sline = -1;
		callStack[height].scolm = -1;
		height++;
		return;
	}

	for (int i=height-1; i>=0; i--) 
		if ( callStack[i].id == caller) {
			height = i+1;
			callStack[i].id = callee;
			callStack[i].sline = -1;
			callStack[i].scolm = -1;
			callStack[i].sline = sline;
			callStack[i].scolm = scolm;
			return;
		}
}

__device__ void* InitKernel(void* ptrhead)
{
	//TODO:
	if ( (blockIdx.x + blockIdx.y*gridDim.x) < CTALB 
		|| (blockIdx.x + blockIdx.y*gridDim.x) > CTAUB) // you only need a few CTAs
		return NULL; 
	int tid = threadIdx.x + threadIdx.y *blockDim.x;
	int bid = blockIdx.x + blockIdx.y * gridDim.x;
	int global_tid = tid + bid*blockDim.x*blockDim.y;
	int num_cta = gridDim.x*gridDim.y;
	int num_thread = blockDim.x*blockDim.y;
	__shared__ char* handler;	//this pointer is for maintaing stack/callpath
	__syncthreads();
	int bytesPerThread = sizeof(CallSite_t)*CALL_PATH_LEN_DEVICE + 32;// I put 32 just to be safe
	if ( tid ==0 ) {
		handler = (char*) malloc( blockDim.x*blockDim.y*bytesPerThread); 
		assert( handler!=NULL);
		int rank = atomicAdd( &aliveCTA, 1);
		printf(" CTA\t%d\tonline, total alive\t%d\n", bid, rank);
		if (rank==0) {
			printf("\nd: InitKernel...\n");
			printf("d: buffer pointer: %p\n", buffer_oN_DeViCe);
			printf("d: size of kernel grid: %d, %d\t%d, %d\n", gridDim.x, gridDim.y, blockDim.x, blockDim.y);
		}
		if (rank == 1)
			buffer_oN_DeViCe = (int*)ptrhead;
	}	
	__syncthreads();
	void* stackzone = (void*)( handler + bytesPerThread*tid );
	return stackzone;
} 

__device__ void callFunc(void* er, void* ee, int sline, int scolm, void* p_stackzone)
{
	int id1 = getFuncID( (char*)er );
	int id2 = getFuncID( (char*)ee );
	int tid = threadIdx.y * blockDim.x + threadIdx.x;
	int bid = blockIdx.x + blockIdx.y * gridDim.x;
	// int global_tid = bid * (blockDim.x * blockDim.y) + tid;
	updateCallStack(id1, id2, (short) sline, (short) scolm, bid, tid, p_stackzone);
}


__device__ void cxtprint(int id)
{
	if (id<0)
		return;
	printf("d::: requested context id: %d out of %d\n", id, cHeight);
	for (int i = 0; i< CALL_PATH_LEN_DEVICE  && contextDic[id][i].id != -1  ; i++)
		printf("d::::::: current context [%d][%d]: %d, %d, %d\n", id, i, contextDic[id][i].id, contextDic[id][i].sline, contextDic[id][i].scolm) ;
	return;
}

__device__ void cxtcpy( CallSite_t* dst, CallSite_t* src , int height) //context copy 
{
	int i;
	for( i=0; i< height; i++)
		dst[i] = src[i];

	dst[i].id = -1; //to mark the ending of one context
	return;
}

__device__ bool cxtcmp( CallSite_t* dst, CallSite_t* src, int height)
{
	for( int i=0; i< height; i++)
		if ( dst[i].id == src[i].id ) // && dst[i].id == src[i].id && 	
			continue;
		else
			return false;
	return true;
}

__device__ int getContextID(void* p_stackzone)
{	//shared by all treahds, there are races
	//you can manually to take care of serialization?
	int bid = blockIdx.x + blockIdx.y * gridDim.x; 
	int tid = threadIdx.y * blockDim.x + threadIdx.x;
	CallSite_t* callStack = (CallSite_t*) p_stackzone;
	int bytesPerThread = (CALL_PATH_LEN_DEVICE*sizeof(CallSite_t));
	int* temp = (int*)( (char*)p_stackzone + bytesPerThread+16);	//offset by 8 to be safe, need to be consistent
	int &height = *temp;
	if ( height ==0) //it is possible that call stack is still empty
		return -1;
	if (cHeight==0) { // the first ever context in the dic
		cxtcpy(contextDic[0], callStack, height );
		cHeight=1;
		return 0;
	}

	int i;
	for (i = 0; i<cHeight; i++) {
		if ( cxtcmp( contextDic[i], callStack, height ) ) //yes, found
			return i; 
	}
	cxtcpy(contextDic[i], callStack, height );
	cHeight = i+1;
	return i;	
}

__device__ void passBasicBlock(int tmp /*pointer to block name*/, int action, int sline, int scolm, void* p_stackzone)
{
	if ( (blockIdx.x + blockIdx.y*gridDim.x) < CTALB 
		|| (blockIdx.x + blockIdx.y*gridDim.x) > CTAUB) // you only need a few CTAs
		return;
	int map = __ballot(1);
	int numActive = __popc(map);
	if ( buffer_oN_DeViCe == NULL)
		return;
	if (numActive == 32) {
		//then choose one thread to write numbers
		int tid = threadIdx.x + threadIdx.y *blockDim.x;
		if (tid%32 == 0) {
			//do the writing
			int bid = atomicAdd(&bbccnntt, 1);
			unsigned long long key=0;
			BBlog_t* bblog = (BBlog_t*) buffer_oN_DeViCe;
			bblog[bid].key = key;
			bblog[bid].tidx = (short)threadIdx.x;
			bblog[bid].tidy = (short)threadIdx.y;
			bblog[bid].bidx = (short)blockIdx.x;
			bblog[bid].bidy = (short)blockIdx.y;
			bblog[bid].sline = sline;
			bblog[bid].scolm = scolm;
			bblog[bid].cid = getContextID(p_stackzone);
		}
	} else {
		//every thread needs to write
		int bid = atomicAdd(&bbccnntt, 1);
		unsigned long long key=0;
		BBlog_t* bblog = (BBlog_t*) buffer_oN_DeViCe;
		bblog[bid].key = key;
		bblog[bid].tidx = (short)threadIdx.x;
		bblog[bid].tidy = (short)threadIdx.y;
		bblog[bid].bidx = (short)blockIdx.x;
		bblog[bid].bidy = (short)blockIdx.y;
		bblog[bid].sline = sline;
		bblog[bid].scolm = scolm;
		bblog[bid].cid = getContextID(p_stackzone);
	}
	return;
}

__device__ void storeLines(void* p, short size/*bytes*/, short line, short colmn, short op /*load or store*/, void* p_stackzone)
{
	if ( (blockIdx.x + blockIdx.y*gridDim.x) < CTALB 
		|| (blockIdx.x + blockIdx.y*gridDim.x) > CTAUB) // you only need a few CTAs
			return;
	int map = __ballot(1);
	int numActive = __popc(map);
	if ( ccnntt >  (int)(((long)BUFFERSIZE)/24) - 128*100)
		return; //DEBUG
	assert ( (ccnntt < BUFFERSIZE/24 - 128) && "code: e31: too many entries to the buffer"); //DO NOT COMMENT OUT
	int bid = atomicAdd(&ccnntt, 1);
	if (buffer_oN_DeViCe==NULL)
		return;
	if( true) {
		int tid = threadIdx.x + threadIdx.y *blockDim.x;
		if ( tid%32==0 || true) {
			short* buffer_oN_DeViCe_short = (short*) buffer_oN_DeViCe;
			long* buffer_oN_DeViCe_long = (long*) buffer_oN_DeViCe;
			buffer_oN_DeViCe_short[bid*12+0] = (short)blockIdx.x;
			buffer_oN_DeViCe_short[bid*12+1] = (short)blockIdx.y;
			buffer_oN_DeViCe_short[bid*12+2] = (short)threadIdx.x;
			buffer_oN_DeViCe_short[bid*12+3] = (short)threadIdx.y;

			buffer_oN_DeViCe_long[bid*3+1] = (long)p;

			buffer_oN_DeViCe_short[bid*12+8] = size;
			buffer_oN_DeViCe_short[bid*12+9] = line;
			buffer_oN_DeViCe_short[bid*12+10] = colmn;
			buffer_oN_DeViCe_short[bid*12+11] = op;
			getContextID(p_stackzone);
		}	
	}
}

__device__ void print1(int a)
{
	if (threadIdx.x + threadIdx.y + blockIdx.x + blockIdx.y == 0)
		printf("d: print1: %d\n", a);

	return;
	if (threadIdx.x + threadIdx.y + blockIdx.x + blockIdx.y == 0 && VERBOSE) {
		if (a==1)
			printf("d: load by CTA (%d,%d)\n", blockIdx.x, blockIdx.y);
		else if (a==2)
			printf("d: store by CTA (%d,%d)\n", blockIdx.x, blockIdx.y);
		else
			printf("d: !!! undefined !!! \n" );
	}	
}

__device__ void print3(int line, int col)
{
	return;
	if (threadIdx.x + threadIdx.y + blockIdx.x + blockIdx.y == 0 && VERBOSE)
		printf("d: source line: %d\t column: %d by CTA (%d,%d)\n", line, col, blockIdx.x, blockIdx.y);
}

__device__ void print4(void* p)
{
	printf("d: print4: %p\n", p);
}

__device__ void print5(void* p, int bits, int sline, int scolm, int op, void* p_stackzone)
{
	if ( (blockIdx.x + blockIdx.y*gridDim.x) < CTALB 
		|| (blockIdx.x + blockIdx.y*gridDim.x) > CTAUB) // you only need a few CTAs 
		return;
	storeLines(p, (short)(bits/8), (short)sline, (short) scolm, (short)op, p_stackzone);
}

__device__ void RetKernel(void* p_stackzone)
{
	if ( (blockIdx.x + blockIdx.y*gridDim.x) < CTALB || (blockIdx.x + blockIdx.y*gridDim.x) > CTAUB) // you only need a few CTAs 
		return;

	int bid = blockIdx.x + blockIdx.y * gridDim.x;
	int tid = threadIdx.x + threadIdx.y *blockDim.x;
	__syncthreads();	//IMPORTANT to sync here
	int rank = -1;
	if ( tid == 0) {
		if (p_stackzone!=NULL) {
			free(p_stackzone);
			rank = atomicAdd( &aliveCTA, -1);
			printf("CTA\t%d\texits, total remains\t%d\n", bid, rank);
		} else
			printf("d:: p_stack is hacked!!\n");
	}
	__syncthreads();
	if (threadIdx.x + threadIdx.y == 0 && rank ==1 ) {
		printf("d: in RetKernel...\n");
		if (true) {	//memory
			short* buffer_oN_DeViCe_short = (short*) buffer_oN_DeViCe;
			buffer_oN_DeViCe_short[0+0] = blockDim.x; // Be consistent with print.cpp, dumpTrace()
			buffer_oN_DeViCe_short[0+1] = blockDim.y;
			buffer_oN_DeViCe_short[0+2] = gridDim.x;
			buffer_oN_DeViCe_short[0+3] = gridDim.y;
			printf("d: Kernel Returns: collected [ %llu ] memory entries. \n" , ccnntt);
			printf("d: Kernel Returns: collected [ %llu ] memory entries. \n" , bbccnntt);

			long* buffer_oN_DeViCe_long = (long*) buffer_oN_DeViCe;
			buffer_oN_DeViCe_long[0+1] = ccnntt;
		} else {	//branch
			BBlog_t* bbbuffer_oN_DeViCe_short = (BBlog_t*) buffer_oN_DeViCe;
			bbbuffer_oN_DeViCe_short[0].bidx = blockDim.x; // Be consistent with print.cpp, dumpTrace()
			bbbuffer_oN_DeViCe_short[0].bidy = blockDim.y;
			bbbuffer_oN_DeViCe_short[0].tidx = gridDim.x;
			bbbuffer_oN_DeViCe_short[0].tidy = gridDim.y;
			bbbuffer_oN_DeViCe_short[0].key = bbccnntt;
			bbbuffer_oN_DeViCe_short[0].sline = 0;
			bbbuffer_oN_DeViCe_short[0].scolm = 0;
			printf("d: Kernel Returns: collected [ %llu ] BB logs. \n" , bbccnntt);
			printf("d: Kernel Returns: collected [ %llu ] BB logs. \n" , ccnntt);
		}
		unsigned long offset1 = ((UNIQUE_FUNC_DEVICE* FUNC_NAME_LEN*sizeof(char))/1024+1)*1024;
		unsigned long offset2 = ((TOTAL_NUMBER_CONTEXT * CALL_PATH_LEN_DEVICE* sizeof(CallSite_t))/1024+1)*1024 + offset1;

		printf("size of function dic: %d %d %lu -> %lu , rounded to %lu\n", UNIQUE_FUNC_DEVICE, FUNC_NAME_LEN, sizeof(char), UNIQUE_FUNC_DEVICE*FUNC_NAME_LEN*sizeof(char), offset1 );
		printf("size of context dic: %d %d %lu -> %lu , rounded to %lu\n", TOTAL_NUMBER_CONTEXT, CALL_PATH_LEN_DEVICE, sizeof(CallSite_t), TOTAL_NUMBER_CONTEXT* CALL_PATH_LEN_DEVICE* sizeof(CallSite_t) , offset2);

		//function dic is the last, 
		//context dic is second to last
		void* ptr;
		ptr = (void*)( buffer_oN_DeViCe + (BUFFERSIZE - offset1)/sizeof(int)) ; //operate on a int*, not a void*
		memcpy( ptr, funcDic, UNIQUE_FUNC_DEVICE *FUNC_NAME_LEN*sizeof(char) );
		ptr = (void*)(buffer_oN_DeViCe + (BUFFERSIZE - offset2)/sizeof(int)) ; //operate on a int*, not a void*
		memcpy( ptr, contextDic, TOTAL_NUMBER_CONTEXT * CALL_PATH_LEN_DEVICE*sizeof(CallSite_t) );

		bbccnntt = 1; //reset, prepares for next kernel call

	}//end of if
}